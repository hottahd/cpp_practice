#include "hip/hip_runtime.h"
#include "advection.cuh"

#include <vector>
#include <cstdlib>
#include <iostream>
#include <cstdio>

#include "grid.cuh"
#include "quantity.cuh"


template<typename Real>
__global__ void update_kernel(Real* q0, Real* q1, Grid_Device<Real> grid, Real dt, Real vc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=1 && i <= grid.i_total-2) {
        q1[i] = q0[i] - vc*(q0[i + 1] - q0[i - 1]) / (2.0 * grid.dx) * dt;
    }
}

template<typename Real>
__global__ void bc_kernel(Real* qq, Grid_Device<Real> grid) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < grid.margin) {
        qq[i] = qq[grid.i_total - 2*grid.margin + i];
        qq[grid.i_total - grid.margin + i] = qq[grid.margin + i];
    }
}

template<typename Real>
__global__ void final_kernel(Real* q0, Real* q2, Grid_Device<Real> grid) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < grid.i_total) {
        q0[i] = 0.5*(q0[i] + q2[i]);
    }
}

template <typename Real>
void Advection<Real>::cfl_condition() {
    Real safety = 0.5; // safety factor

    time.dt = 1.e10;
    for (int i = 0; i < grid.i_total; ++i) {
        time.dt = std::min(time.dt, safety * grid.dx / std::abs(vc));
    }
};

template <typename Real>
void Advection<Real>::run() {

    cfl_condition();

    quantity.allocate_device(grid);
    quantity.upload_q0();
    
    while (time.time < time.tend) {
        update();
        time.update();
        io_step();
    };
    quantity.free_device();

}


template <typename Real>
void Advection<Real>::update() {
    sc2ssprk();
}

template <typename Real>
void Advection<Real>::io_step() {
    if (time.time >= time.n_output * time.dt_output) {
        quantity.download_q0();
        quantity.save(config, time);
        std::cout
        << "time = " 
        << std::fixed << std::setprecision(2) << std::setw(8) << time.time
        << ";  n_step = " 
        << std::setw(8) << time.n_step
        << ";  n_output = " 
        << std::setw(8) << time.n_output
        << std::endl;

        time.n_output++;
    }
}


template <typename Real>
void Advection<Real>::sc2ssprk() {
    
    int block_size = 256;
    int grid_size = (grid.i_total + block_size - 1) / block_size;
    update_kernel<<<grid_size, block_size>>>(quantity.q0_dev, quantity.q1_dev, grid_device, time.dt, vc);
    bc_kernel<<<grid_size, block_size>>>(quantity.q1_dev, grid_device);

    update_kernel<<<grid_size, block_size>>>(quantity.q1_dev, quantity.q2_dev, grid_device, time.dt, vc);
    bc_kernel<<<grid_size, block_size>>>(quantity.q2_dev, grid_device);
    
    final_kernel<<<grid_size, block_size>>>(quantity.q0_dev, quantity.q2_dev, grid_device);

}

template<typename Real>
void Advection<Real>::bc(std::vector<Real>& qq, Grid<Real>& grid) {
    for (int i = 0; i < grid.margin; ++i) {
        qq[i] = qq[grid.i_total - 2*grid.margin + i];
        qq[grid.i_total - grid.margin + i] = qq[grid.margin + i];
    }
}

template struct Advection<double>;
template struct Advection<float>;
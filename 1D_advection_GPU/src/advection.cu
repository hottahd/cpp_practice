#include "hip/hip_runtime.h"
#include "advection.cup"
#include "grid.hpp"
#include "iostream"
#include "quantity.hpp"
#include <vector>
#include <cstdlib>
#include <iostream>
#include <cstdio>

template<typename Real>
__device__ void update_core_kernel(Real* q0, Real* q1, Real dx, Real dt, Real vc, int i) {
    q1[i] = q0[i] - vc*(q0[i + 1] - q0[i - 1]) / (2.0 * dx) * dt;
}

template<typename Real>
__global__ void update_kernel(Real* q0, Real* q1, int i_total, Real dx, Real dt, Real vc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=1 && i <= i_total-2) {
        update_core_kernel(q0, q1, dx, dt, vc, i);
    }
}

template<typename Real>
__global__ void bc_kernel(Real* qq, int i_total, int margin) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < margin) {
        qq[i] = qq[i_total - 2*margin + i];
        qq[i_total - margin + i] = qq[margin + i];
    }
}

template <typename Real>
void Advection<Real>::cfl_condition() {
    Real safety = 0.5; // safety factor

    time.dt = 1.e10;
    for (int i = 0; i < grid.i_total; ++i) {
        time.dt = std::min(time.dt, safety * grid.dx / std::abs(vc));
    }
};

template <typename Real>
void Advection<Real>::update() {
    cfl_condition();
    sc2ssprk();
}

template <typename Real>
void Advection<Real>::io_step() {
    if (time.time >= time.n_output * time.dt_output) {
        quantity.save(config, time);
        std::cout
        << "time = " 
        << std::fixed << std::setprecision(2) << std::setw(8) << time.time
        << ";  n_step = " 
        << std::setw(8) << time.n_step
        << ";  n_output = " 
        << std::setw(8) << time.n_output
        << std::endl;

        time.n_output++;
    }
}


template <typename Real>
void Advection<Real>::sc2ssprk() {

    quantity.allocate_device(grid);
    quantity.upload_q0();
    
    int block_size = 256;
    int grid_size = (grid.i_total + block_size - 1) / block_size;
    update_kernel<<<grid_size, block_size>>>(quantity.q0_dev, quantity.q1_dev, grid.i_total, grid.dx, time.dt, vc);
    bc_kernel<<<grid_size, block_size>>>(quantity.q1_dev, grid.i_total, grid.margin);

    update_kernel<<<grid_size, block_size>>>(quantity.q1_dev, quantity.q2_dev, grid.i_total, grid.dx, time.dt, vc);
    bc_kernel<<<grid_size, block_size>>>(quantity.q2_dev, grid.i_total, grid.margin);
    hipDeviceSynchronize();

    hipMemcpy(quantity.q1.data(), quantity.q1_dev, sizeof(Real) * grid.i_total, hipMemcpyDeviceToHost);
    hipMemcpy(quantity.q2.data(), quantity.q2_dev, sizeof(Real) * grid.i_total, hipMemcpyDeviceToHost);
    
    for ( int i = 0; i < grid.i_total; ++i) {
        quantity.q0[i] = 0.5*( quantity.q0[i] + quantity.q2[i] );
    }

    quantity.free_device();

}

// template<typename Real>
// void Advection<Real>::sc2(std::vector<Real>& qq, std::vector<Real>& dqq, Grid<Real>& grid, Real& dt, Real& vc) {

//     for (int i = 1; i < grid.i_total - 1; ++i) {
//         dqq[i] = -vc * (qq[i + 1] - qq[i - 1]) / (2.0 * grid.dx)*dt;
//     };
// }

template<typename Real>
void Advection<Real>::bc(std::vector<Real>& qq, Grid<Real>& grid) {
    for (int i = 0; i < grid.margin; ++i) {
        qq[i] = qq[grid.i_total - 2*grid.margin + i];
        qq[grid.i_total - grid.margin + i] = qq[grid.margin + i];
    }
}

template struct Advection<double>;
template struct Advection<float>;
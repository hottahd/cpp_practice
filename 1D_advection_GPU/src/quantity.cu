#include "hip/hip_runtime.h"
#include "quantity.hpp"
#include "utility.hpp"
#include "time.hpp"
#include <cmath>

template<typename Real>
void Quantity<Real>::allocate_device(const Grid<Real>& grid) {
    hipMalloc(&q0_dev,  sizeof(Real) * grid.i_total);
    hipMalloc(&q1_dev,  sizeof(Real) * grid.i_total);
    hipMalloc(&q2_dev,  sizeof(Real) * grid.i_total);
    hipMalloc(&qq_dev,  sizeof(Real) * grid.i_total);
    hipMalloc(&dqq_dev, sizeof(Real) * grid.i_total);
}

template<typename Real>
void Quantity<Real>::upload_q0() {
    hipMemcpy(q0_dev, q0.data(), sizeof(Real) * q0.size(), hipMemcpyHostToDevice);
}

template<typename Real>
void Quantity<Real>::download_q0() {
    hipMemcpy(q0.data(), q0_dev, sizeof(Real) * q0.size(), hipMemcpyDeviceToHost);
}

template<typename Real>
void Quantity<Real>::free_device() {
    hipFree(q1_dev);
    hipFree(q2_dev);
    hipFree(qq_dev);
    hipFree(dqq_dev);
}

template <typename Real>
void Quantity<Real>::initial_condition(const Grid<Real>& grid) {
    Real dd = 0.1; // width of Gaussian 
    Real xm = 0.5*(grid.xmax + grid.xmin); // center of Gaussian
    
    for (int i = 0; i < grid.i_total; ++i) {
        q0[i] = std::exp(-std::pow((grid.x[i] - xm) / dd, 2));
    }
}

template <typename Real>
void Quantity<Real>::save(const Config& config, const Time<Real>& time) const {
    std::ofstream ofs(config.save_dir + "/quantity."+ util::zfill(time.n_output, time.n_output_digits) +".bin", std::ios::binary);
    assert(ofs.is_open());
    ofs.write(reinterpret_cast<const char*>(q0.data()), sizeof(Real)*q0.size());
}

template struct Quantity<double>;
template struct Quantity<float>;